/**
 * @file cli.cu
 * @author Jiannan Tian
 * @brief
 * @version 0.3
 * @date 2022-03-07
 *
 * (C) 2022 by Washington State University, Argonne National Laboratory
 *
 */

#include "cli.cuh"

template class cusz::CLI<float>;
