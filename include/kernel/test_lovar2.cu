/**
 * @file test_lorenzo_var.cu
 * @author Jiannan Tian
 * @brief
 * @version 0.3
 * @date 2022-09-29
 *
 * (C) 2022 by Indiana University, Argonne National Laboratory
 *
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <thrust/execution_policy.h>
#include <thrust/for_each.h>
#include <iostream>
#include <string>

#include "../cli/quality_viewer.hh"
#include "../utils/cuda_err.cuh"
#include "../utils/io.hh"
#include "lorenzo_var.cuh"

using std::cerr;
using std::cout;
using std::endl;

template <typename DeltaT = uint16_t>
int f(std::string fname, size_t x, size_t y, size_t z, double eb, size_t start = 10000)
{
    float*  h_data;
    float*  data;
    float*  xdata;
    bool*   signum;
    DeltaT* delta;

    dim3   len3    = dim3(x, y, z);
    dim3   stride3 = dim3(1, x, x * y);
    size_t len     = x * y * z;

    hipHostMalloc(&h_data, len * sizeof(float));

    hipMalloc(&data, len * sizeof(float));
    hipMalloc(&xdata, len * sizeof(float));
    hipMalloc(&signum, len * sizeof(bool));
    hipMalloc(&delta, len * sizeof(DeltaT));

    io::read_binary_to_array<float>(fname, h_data, len);
    hipMemcpy(data, h_data, len * sizeof(float), hipMemcpyHostToDevice);

    {
        printf("data\n");
        thrust::for_each(thrust::device, data + start, data + start + 20, [=] __device__ __host__(const float i) {
            printf("%.3e\t", i);
        });
        printf("\n");
    }

    hipStream_t stream;
    hipStreamCreate(&stream);

    float time_comp;

    cusz::experimental::launch_construct_LorenzoI_var<float, DeltaT, float>(
        data, delta, signum, len3, eb, time_comp, stream);

    {
        printf("signum\n");
        thrust::for_each(thrust::device, signum + start, signum + start + 20, [=] __device__ __host__(const bool i) {
            printf("%d\t", (int)i);
        });
        printf("\n");
        printf("delta\n");
        thrust::for_each(thrust::device, delta + start, delta + start + 20, [=] __device__ __host__(const DeltaT i) {
            printf("%u\t", (uint32_t)i);
        });
        printf("\n");
    }

    cout << "comp time\t" << time_comp << endl;

    float time_decomp;
    cusz::experimental::launch_reconstruct_LorenzoI_var<float, DeltaT, float>(
        signum, delta, xdata, len3, eb, time_decomp, stream);

    cout << "decomp time\t" << time_decomp << endl;

    {
        printf("xdata\n");
        thrust::for_each(thrust::device, xdata + start, xdata + start + 20, [=] __device__ __host__(const float i) {
            printf("%.3e\t", i);
        });
        printf("\n");
    }

    /* perform evaluation */ cusz::QualityViewer::echo_metric_gpu(xdata, data, len);

    hipHostFree(h_data);
    hipFree(data);
    hipFree(xdata);
    hipFree(signum);
    hipFree(delta);

    hipStreamDestroy(stream);

    return 0;
}

int main(int argc, char** argv)
{
    if (argc < 5) {
        cout << "                       default: ui16" << endl;
        cout << "                       ui8,ui16,ui32" << endl;
        cout << "PROG fname x y z [eb] [delta type] [print offset]" << endl;
        cout << "0    1     2 3 4 [5]  [6]          [7]" << endl;

        return 1;
    }

    auto fname = std::string(argv[1]);
    auto x     = atoi(argv[2]);
    auto y     = atoi(argv[3]);
    auto z     = atoi(argv[4]);

    double      eb          = 1e-4;
    std::string delta_type  = "ui16";
    size_t      print_start = 10000;

    if (argc >= 6) eb = atof(argv[5]);

    if (argc >= 7) delta_type = std::string(argv[6]);

    if (argc >= 8) print_start = atoi(argv[7]);

    if (delta_type == "ui8")
        f<uint8_t>(fname, x, y, z, eb, print_start);
    else if (delta_type == "ui16")
        f<uint16_t>(fname, x, y, z, eb, print_start);
    else if (delta_type == "ui32")
        f<uint32_t>(fname, x, y, z, eb, print_start);

    return 0;
}