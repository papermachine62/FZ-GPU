#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <dirent.h>
#include <stdint.h>
#include <sys/stat.h>
#include <thrust/copy.h>
#include <chrono>
#include <hipcub/hipcub.hpp>
#include <fstream>
#include <iostream>
#include <algorithm>

#include "include/kernel/lorenzo_var.cuh"
#include "include/utils/cuda_err.cuh"
#include "include/utils/io.hh"

#define UINT32_BIT_LEN 32


long GetFileSize(std::string filename)
{
    struct stat stat_buf;
    int         rc = stat(filename.c_str(), &stat_buf);
    return rc == 0 ? stat_buf.st_size : -1;
}

template <typename T>
T* read_binary_to_new_array(const std::string& fname, size_t dtype_len)
{
    std::ifstream ifs(fname.c_str(), std::ios::binary | std::ios::in);
    if (not ifs.is_open()) {
        std::cerr << "fail to open " << fname << std::endl;
        exit(1);
    }
    auto _a = new T[dtype_len]();
    ifs.read(reinterpret_cast<char*>(_a), std::streamsize(dtype_len * sizeof(T)));
    ifs.close();
    return _a;
}

template <typename T>
void write_array_to_binary(const std::string& fname, T* const _a, size_t const dtype_len)
{
    std::ofstream ofs(fname.c_str(), std::ios::binary | std::ios::out);
    if (not ofs.is_open()) return;
    ofs.write(reinterpret_cast<const char*>(_a), std::streamsize(dtype_len * sizeof(T)));
    ofs.close();
}

// processing 16 bytes * 32 threads = 512 bytes -> 4 * 128 bytes
__global__ void
generateBitFlagArrayDebug(int blockSize, uint8_t* _d_in, uint32_t* d_bitFlagArray, uint8_t* d_byteFlagArray)
{
    __shared__ struct {
        uint32_t databuffer[128];
    } shm;

    static const int WARPSIZE = 32;
    // at the same time, WARPSIZE = blockDim.x

    auto d_in = reinterpret_cast<uint32_t*>(_d_in);

    const auto gidx_base = 128 * blockIdx.x;

    for (auto i = 0; i < 4; i++) {
        auto local_idx            = threadIdx.x + WARPSIZE * i;
        shm.databuffer[local_idx] = d_in[gidx_base + local_idx];
    }
    __syncthreads();

    uint32_t sum = 0;
    for (auto i = 0; i < 4; i++) { sum |= shm.databuffer[i + threadIdx.x * 4]; }

    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    d_byteFlagArray[tid] = (sum != 0);
    auto ballot_res      = __ballot_sync(0xFFFFFFFFU, sum != 0);
    if (threadIdx.x == 0) d_bitFlagArray[blockIdx.x] = ballot_res;
}

__global__ void encodeDebug(int blockSize, uint8_t* d_in, uint8_t* d_out, uint32_t* preSum)
{
    __shared__ uint32_t sumArr[33];
    int                 tid = threadIdx.x + blockIdx.x * blockDim.x;
    sumArr[0]               = preSum[tid];
    sumArr[threadIdx.x + 1] = preSum[tid + 1];
    __syncthreads();
    if (sumArr[threadIdx.x + 1] != sumArr[threadIdx.x]) {
        for (int i = 0; i < blockSize; i++) { d_out[sumArr[threadIdx.x] * blockSize + i] = d_in[tid * blockSize + i]; }
    }
}

__global__ void bitshuffleDebug(const uint32_t* __restrict__ in, uint32_t* __restrict__ out)
{
    /*
    grid 32x32 threads
    each thread loads 4 bytes (aligned) = 128 bytes per row of 32
    total bytes loaded = 32x32x4 = 4096 bytes
                  x                y  z
    blocks = ( total_bytes / 8192, 2, 1 )
    */
    __shared__ uint32_t smem[32][33];
    uint32_t            v;
    /* This thread is going to load 4 bytes. Next thread in x will load
    the next 4 to be aligned. In total we pick up 32*4 = 128 bytes in this
    row of 32 (warp) for bit0.
    The next row (warp) is going to pick up bit1, etc
    The first grid starts at byte 0 + blockIdx.x * 2048
    The second grid starts at byte 8192/32/2
     */
    smem[threadIdx.y][threadIdx.x] =
        in[threadIdx.x +        // Aligned loads. 32*4 = 128 bytes
           threadIdx.y * 32 +   // Offset to next bit = 8192/32/4.
           blockIdx.x * 2048 +  // Start of the block
           blockIdx.y * 1024];  // Next 32 reads
    __syncthreads();            /* Now we loaded 4 kB to smem.   Do the first level of transpose */
    v = smem[threadIdx.y][threadIdx.x];
#pragma unroll 32
    for (int i = 0; i < 32; i++) smem[threadIdx.y][i] = __ballot_sync(0xFFFFFFFFU, v & (1U << i));
    __syncthreads(); /* Now we loaded 4 kB to smem.   Do the first level of transpose */
    out[threadIdx.x + threadIdx.y * 32 + blockIdx.y * 1024 + blockIdx.x * 2048] = smem[threadIdx.x][threadIdx.y];
}

__global__ void bitshuffleAndBitflag(
    const uint32_t* __restrict__ in,
    uint32_t* __restrict__ out,
    int       blockSize,
    uint32_t* d_bitFlagArray,
    uint32_t*  d_byteFlagArray)
{
    /*
    grid 32x32 threads
    each thread loads 4 bytes (aligned) = 128 bytes per row of 32
    total bytes loaded = 32x32x4 = 4096 bytes
                  x                y  z
    blocks = ( total_bytes / 8192, 2, 1 )
    */
    __shared__ uint32_t smem[32][33];
    uint32_t            v;
    /* This thread is going to load 4 bytes. Next thread in x will load
    the next 4 to be aligned. In total we pick up 32*4 = 128 bytes in this
    row of 32 (warp) for bit0.
    The next row (warp) is going to pick up bit1, etc
    The first grid starts at byte 0 + blockIdx.x * 2048
    The second grid starts at byte 8192/32/2
     */
    smem[threadIdx.y][threadIdx.x] =
        in[threadIdx.x +        // Aligned loads. 32*4 = 128 bytes
           threadIdx.y * 32 +   // Offset to next bit = 8192/32/4.
           blockIdx.x * 2048 +  // Start of the block
           blockIdx.y * 1024];  // Next 32 reads
    __syncthreads();            /* Now we loaded 4 kB to smem.   Do the first level of transpose */
    v = smem[threadIdx.y][threadIdx.x];
#pragma unroll 32
    for (int i = 0; i < 32; i++) smem[threadIdx.y][i] = __ballot_sync(0xFFFFFFFFU, v & (1U << i));
    __syncthreads(); /* Now we loaded 4 kB to smem.   Do the first level of transpose */
    out[threadIdx.x + threadIdx.y * 32 + blockIdx.y * 1024 + blockIdx.x * 2048] = smem[threadIdx.x][threadIdx.y];

    __shared__ uint32_t bitflagArr[8];
    __shared__ uint32_t  byteFlagArray[256];
    if (threadIdx.x * 4 < 32) {
        for (int i = 1; i < 4; i++) { smem[threadIdx.x * 4][threadIdx.y] |= smem[threadIdx.x * 4 + i][threadIdx.y]; }
        byteFlagArray[threadIdx.y * 8 + threadIdx.x] = (smem[threadIdx.x * 4][threadIdx.y] > 0);
    }
    __syncthreads();
    uint32_t buffer;
    if (threadIdx.y < 8) {
        buffer                  = byteFlagArray[threadIdx.y * 32 + threadIdx.x];
        bitflagArr[threadIdx.y] = __ballot_sync(0xFFFFFFFFU, buffer);
    }
    __syncthreads();
    if (threadIdx.y < 8) {
        d_byteFlagArray[blockIdx.x * 512 + blockIdx.y * 256 + threadIdx.y * 32 + threadIdx.x] =
            byteFlagArray[threadIdx.y * 32 + threadIdx.x];
    }
    if (threadIdx.x < 8 && threadIdx.y == 0) {
        d_bitFlagArray[blockIdx.x * 16 + blockIdx.y * 8 + threadIdx.x] = bitflagArr[threadIdx.x];
    }
}

__global__ void halfBitshuffleDebug(const uint32_t* __restrict__ in, uint32_t* __restrict__ out)
{
    /*
    grid 32x32 threads
    each thread loads 4 bytes (aligned) = 128 bytes per row of 32
    total bytes loaded = 32x32x4 = 4096 bytes
                  x                y  z
    blocks = ( total_bytes / 8192, 2, 1 )
    */
    __shared__ uint32_t smem[32][33];
    uint32_t            v;
    /* This thread is going to load 4 bytes. Next thread in x will load
    the next 4 to be aligned. In total we pick up 32*4 = 128 bytes in this
    row of 32 (warp) for bit0.
    The next row (warp) is going to pick up bit1, etc
    The first grid starts at byte 0 + blockIdx.x * 2048
    The second grid starts at byte 8192/32/2
     */
    smem[threadIdx.y][threadIdx.x] =
        in[threadIdx.x +        // Aligned loads. 32*4 = 128 bytes
           threadIdx.y * 32 +   // Offset to next bit = 8192/32/4.
           blockIdx.x * 2048 +  // Start of the block
           blockIdx.y * 1024];  // Next 32 reads
    __syncthreads();            /* Now we loaded 4 kB to smem.   Do the first level of transpose */
    v = smem[threadIdx.y][threadIdx.x];
#pragma unroll 8
    for (int i = 0; i < 8; i++) smem[threadIdx.y][i] = __ballot_sync(0xFFFFFFFFU, v & (1U << i));
#pragma unroll 8
    for (int i = 16; i < 24; i++) smem[threadIdx.y][i] = __ballot_sync(0xFFFFFFFFU, v & (1U << i));
    int     quotient                                            = threadIdx.x / 4;
    int     reminder                                            = threadIdx.x % 4;
    uint8_t first                                               = *((uint8_t*)(&v));
    uint8_t third                                               = *((uint8_t*)(&v) + 2);
    *((uint8_t*)(&smem[threadIdx.y][8 + quotient]) + reminder)  = first;
    *((uint8_t*)(&smem[threadIdx.y][24 + quotient]) + reminder) = third;
    __syncthreads(); /* Now we loaded 4 kB to smem.   Do the first level of transpose */
    out[threadIdx.x + threadIdx.y * 32 + blockIdx.y * 1024 + blockIdx.x * 2048] = smem[threadIdx.x][threadIdx.y];
}

void runSzs(std::string fileName, int x, int y, int z, double eb)
{
    // auto len3 = dim3(3600, 1800, 1);
    auto len3     = dim3(x, y, z);
    int  fileSize = GetFileSize(fileName);
    auto len      = int(fileSize / sizeof(float));

    float*    data;
    float*    h_data;
    bool*     signum;
    uint16_t* quant;
    float     time_elapsed;


    h_data = read_binary_to_new_array<float>(fileName, len);
    float range = *std::max_element(h_data , h_data + len) - *std::min_element(h_data , h_data + len);
    CHECK_CUDA(hipMalloc((void**)&data, sizeof(float) * len));
    CHECK_CUDA(hipMemcpy(data, h_data, sizeof(float) * len, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMalloc((void**)&quant, sizeof(uint16_t) * len));
    CHECK_CUDA(hipMemset(quant, 0, sizeof(uint16_t) * len));
    CHECK_CUDA(hipMalloc((void**)&signum, sizeof(bool) * len));

    hipStream_t stream;
    hipStreamCreate(&stream);

    // absolute error bound
    cusz::experimental::launch_construct_LorenzoI_var<float, uint16_t, float>(
        data, quant, signum, len3, eb * range, time_elapsed, stream);

    CHECK_CUDA(hipFree(data));
    CHECK_CUDA(hipFree(signum));

    uint16_t* bitshuffleOut;
    CHECK_CUDA(hipMalloc((void**)&bitshuffleOut, sizeof(uint16_t) * len));
    CHECK_CUDA(hipMemcpy(bitshuffleOut, quant, sizeof(uint16_t) * len, hipMemcpyDeviceToDevice));

    int  blockSize    = 16;
    auto newLen       = len * 2;  // bitshuffle result length in byte unit
    newLen            = newLen % 8192 == 0 ? newLen : newLen - newLen % 8192 + 8192;
    int dataChunkSize = newLen % (blockSize * UINT32_BIT_LEN) == 0 ? newLen / (blockSize * UINT32_BIT_LEN)
                                                                   : int(newLen / (blockSize * UINT32_BIT_LEN)) + 1;
    uint32_t* d_bitFlagArray;
    uint32_t*  d_byteFlagArray;

    uint8_t* d_out;
    CHECK_CUDA(hipMalloc((void**)&d_bitFlagArray, sizeof(uint32_t) * dataChunkSize));
    CHECK_CUDA(hipMemset(d_bitFlagArray, 0, sizeof(uint32_t) * dataChunkSize));
    CHECK_CUDA(hipMalloc((void**)&d_byteFlagArray, sizeof(uint32_t) * dataChunkSize * UINT32_BIT_LEN));
    CHECK_CUDA(hipMemset(d_byteFlagArray, 0, sizeof(uint32_t) * dataChunkSize * UINT32_BIT_LEN));

    CHECK_CUDA(hipMalloc((void**)&d_out, sizeof(uint8_t) * dataChunkSize * blockSize * UINT32_BIT_LEN));
    CHECK_CUDA(hipMemset(d_out, 0, sizeof(uint8_t) * dataChunkSize * blockSize * UINT32_BIT_LEN));

    dim3 threads(32, 32);
    dim3 grid(floor(newLen / 8192), 2, 1);  // divided by 2 is because the file is transformed from uint32 to uint16
    bitshuffleAndBitflag<<<grid, threads>>>(
        (uint32_t*)quant, (uint32_t*)bitshuffleOut, blockSize, d_bitFlagArray, d_byteFlagArray);
    CHECK_CUDA(hipFree(quant));

    newLen        = len * 2;  // bitshuffle result length in byte unit
    dataChunkSize = newLen % (blockSize * UINT32_BIT_LEN) == 0 ? newLen / (blockSize * UINT32_BIT_LEN)
                                                               : int(newLen / (blockSize * UINT32_BIT_LEN)) + 1;
    uint32_t* d_preSumArray;
    CHECK_CUDA(
        hipMalloc((void**)&d_preSumArray, sizeof(uint32_t) * dataChunkSize * UINT32_BIT_LEN + sizeof(uint32_t)));
    CHECK_CUDA(hipMemset(d_preSumArray, 0, sizeof(uint32_t) * dataChunkSize * UINT32_BIT_LEN + sizeof(uint32_t)));
    uint32_t* d_byteFlagArrayTest;
    CHECK_CUDA(hipMalloc((void**)&d_byteFlagArrayTest, sizeof(uint32_t) * dataChunkSize * UINT32_BIT_LEN));
    CHECK_CUDA(hipMemcpy(
        d_byteFlagArrayTest, d_byteFlagArray, sizeof(uint32_t) * dataChunkSize * UINT32_BIT_LEN,
        hipMemcpyDeviceToDevice));
    CHECK_CUDA(hipFree(d_byteFlagArray));

    void*  d_temp_storage     = NULL;
    size_t temp_storage_bytes = 0;
    hipcub::DeviceScan::ExclusiveSum(
        d_temp_storage, temp_storage_bytes, d_byteFlagArrayTest, d_preSumArray, dataChunkSize * UINT32_BIT_LEN);
    // Allocate temporary storage
    CHECK_CUDA(hipMalloc(&d_temp_storage, temp_storage_bytes));
    // Run exclusive prefix sum
    hipcub::DeviceScan::ExclusiveSum(
        d_temp_storage, temp_storage_bytes, d_byteFlagArrayTest, d_preSumArray, dataChunkSize * UINT32_BIT_LEN);
    
    uint32_t* lastSum = (uint32_t*)malloc(sizeof(uint32_t));
    CHECK_CUDA(
        hipMemcpy(lastSum, d_preSumArray + dataChunkSize * UINT32_BIT_LEN - 1, sizeof(uint32_t), hipMemcpyDeviceToHost));
    uint32_t* lastFlag = (uint32_t*)malloc(sizeof(uint32_t));
    CHECK_CUDA(
        hipMemcpy(lastFlag, d_byteFlagArrayTest + dataChunkSize * UINT32_BIT_LEN - 1, sizeof(uint32_t), hipMemcpyDeviceToHost));
    uint32_t* result = (uint32_t*)malloc(sizeof(uint32_t));
    *result = *lastSum + *lastFlag;
    CHECK_CUDA(
        hipMemcpy(d_preSumArray + dataChunkSize * UINT32_BIT_LEN, result, sizeof(uint32_t), hipMemcpyHostToDevice));
    CHECK_CUDA(hipFree(d_byteFlagArrayTest));
    CHECK_CUDA(hipFree(d_temp_storage));
    encodeDebug<<<dataChunkSize, 32>>>(
        blockSize, (uint8_t*)bitshuffleOut, d_out, d_preSumArray);  
    printf("original size: %d\n", fileSize);
    printf("compressed size: %d\n", sizeof(uint32_t) * dataChunkSize + blockSize * (*result));
    printf(
        "compression ratio: %f\n", float(fileSize) / float(sizeof(uint32_t) * dataChunkSize + blockSize * (*result)));
    hipStreamDestroy(stream);
    CHECK_CUDA(hipFree(bitshuffleOut));
    CHECK_CUDA(hipFree(d_bitFlagArray));
    CHECK_CUDA(hipFree(d_preSumArray));
    CHECK_CUDA(hipFree(d_out));
    delete[] h_data;
    free(result);
    free(lastSum);
    free(lastFlag);
    return;
}

int main(int argc, char* argv[])
{
    using T = float;
    std::string fileName;
    fileName  = std::string(argv[1]);
    int    x  = std::stoi(argv[2]);
    int    y  = std::stoi(argv[3]);
    int    z  = std::stoi(argv[4]);
    double eb = std::stod(argv[5]);

    runSzs(fileName, x, y, z, eb);
    return 0;
}